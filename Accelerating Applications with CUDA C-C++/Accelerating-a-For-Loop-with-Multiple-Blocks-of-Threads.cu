
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This idiomatic expression gives each thread
 * a unique index within the entire grid.
 */

__global__ void loop()
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%d\n", i);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   */

  loop<<<2, 5>>>();
  hipDeviceSynchronize();
}
