
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 10000;
  int *a;
  
  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);
  
  init(a, N);
  
  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;
  
  /*
   * Initialization of variables to detect synchronous and asynchronous errors in 
   * the codeInitialization of variables to detect synchronous
   * and asynchronous errors in the code.
   */
  hipError_t errSync, errAsync;
  
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  
  /*
   * Catch errors for both the kernel launch above and any
   * errors that occur during the asynchronous `doubleElements`
   * kernel execution.
   */
  
  errSync = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  
  /*
   * Print errors should they exist.
   */
  
  if(errSync != hipSuccess){
      printf("Error: %s\n", hipGetErrorString(errSync));
  }   
  if(errAsync != hipSuccess){
      printf("Error: %s\n", hipGetErrorString(errAsync));
  }

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
