
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * The new kernel should only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
    printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * Refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   */

  loop<<<1, 10>>>();
  hipDeviceSynchronize();
}
