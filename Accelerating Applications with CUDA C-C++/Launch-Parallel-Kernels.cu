
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * The function below runs on the GPU.
 */

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  /*
   * firstParallel execute in parallel on the GPU.
   */

  firstParallel<<<5, 5>>>();

  /*
   * Kernel synchronization
   */
   hipDeviceSynchronize();

}
