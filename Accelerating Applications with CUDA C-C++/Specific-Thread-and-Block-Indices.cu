
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  } 
}

int main()
{
  /*
   * Configuration so that the kernel will print `"Success!"`.
   */

  printSuccessForCorrectExecutionConfiguration<<<256, 1024>>>();
  
  /*
   * kernel execution is asynchronous and you must sync on its completion
   */
  hipDeviceSynchronize();
}
