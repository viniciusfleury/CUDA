
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * The function below runs on the GPU.
 */

__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{

  helloCPU();

  /*
   * This function is called so that it starts as a kernel on the GPU.
   */

  helloGPU<<<1, 1>>>();

  /*
   * kernel synchronization
   */
   hipDeviceSynchronize();
}
