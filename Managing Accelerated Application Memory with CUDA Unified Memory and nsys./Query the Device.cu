
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  /*
   * Assign values to these variables so that the output string below prints the
   * requested properties of the currently active GPU.
   */

  int deviceId;
  hipGetDevice(&deviceId);
  
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  
  
  int computeCapabilityMajor;
  int computeCapabilityMinor;
  int multiProcessorCount;
  int warpSize; 
  
  computeCapabilityMajor = props.major;
  computeCapabilityMinor = props.minor;
  multiProcessorCount = props.multiProcessorCount;
  warpSize = props.warpSize;
  

  /*
   * There should be no need to modify the output string below.
   */

  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}
