
#include <hip/hip_runtime.h>
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);
  
  /*size_t threadsPerBlock = 256;
   *size_t numberOfBlocks = (N + threadsPerBlock - 1)/threadsPerBlock;
   */
  
  hostFunction(a, N);

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU?
   * What happens when unified memory is accessed only by the CPU?
   * What happens when unified memory is accessed first by the GPU then the CPU?
   * What happens when unified memory is accessed first by the CPU then the GPU?
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiment, and then verify by running `nsys`.
   */

  hipFree(a);
}
